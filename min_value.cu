#include<stdio.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

__global__ void max_value(int *a, int *b) // kernel subroutine called from the host cpu to the gpu device
{
	int i = threadIdx.x; // getting the thread id 

	*b = a[0]; // equating the value of the address in c to a predefined value in a matrix.

	if(a[i] < *b) // getting the value from the address of c variable.
	{
		*b = a[i];
	}
}


int main(int argc, char *argv[])
{
	if(argc == 3)
	{
		int arr_size = atoi(argv[1]);
		int rem = atoi(argv[2]);
		srand(time(NULL)); // seed entry for max randomness
		int a[arr_size];
		int b;
		int *dev_a,*dev_b; // gpu parameters
		hipMalloc((void **) &dev_a,arr_size*sizeof(int));
		hipMalloc((void **) &dev_b,sizeof(int));

		for(int i=0;i<arr_size;i++)
		{
			a[i] = rand() % rem + i;
		}
		hipMemcpy(dev_a,a,arr_size*sizeof(int),hipMemcpyHostToDevice); // sending the array into gpu memory.
		max_value<<<1,arr_size>>>(dev_a,dev_b);
		hipMemcpy(&b,dev_b,sizeof(int),hipMemcpyDeviceToHost);
	    printf("\nmax =  %d ",b);
		hipFree(dev_a);		// Free the allocated memory
		hipFree(dev_b);
	}
	
	else if(argc > 3){
			printf("Too many args to be parsed\n");
	}
	else
	{
			printf("one argument required\n");
	}
	return 0;
}