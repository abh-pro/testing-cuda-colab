#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__ void hello()
{
	printf("GPU:: Hello world!!\n");
}

int main()
{
	hello<<<1,10>>>();
	hipDeviceSynchronize();
	return 0;
}