#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__void hello()
{
	printf("GPU:: Hello world!!");
}

int main()
{
	hello<<1,10>>();
	hipDeviceSynchronize();
	return 0;
}